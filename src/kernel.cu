#include "hip/hip_runtime.h"
#define GPU_HOST_ATTR __device__ __host__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>


#include "types.h"



namespace gpusat {


__device__ uint64_t atomicAdd(uint64_t* address, uint64_t val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (val + (uint64_t)(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return old;
}

__device__ uint64_t atomicSub(uint64_t* address, uint64_t val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        ((uint64_t)(assumed) - val));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return old;
}

__device__ int64_t atomicMax(int64_t* address, int64_t val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (max(val, (int64_t)(assumed))));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    return old;
}

// FIXME: normal atomicAdd might not be atomic across devices


__device__ int64_t get_global_id(const RunMeta& meta) {
    // TODO: y and z
    int64_t id = blockDim.x * blockIdx.x + threadIdx.x + meta.minId;
    if (id >= meta.maxId) {
        return -1;
    }
    return id;
}

/**
 * returns the model count which corresponds to the given id
 *
 * @param id
 *      the id for which the model count should be returned
 * @param tree
 *      a pointer to the tree structure
 * @param numVars
 *      the number of variables in the bag
 * @return
 *      the model count
 */
__host__ __device__ double getCount(int64_t id, const TreeNode *tree, long numVars) {
    ulong nextId = 0;
    for (ulong i = 0; i < numVars; i++) {
        nextId = ((uint32_t *) &(tree[nextId]))[(id >> (numVars - i - 1)) & 1];
        if (nextId == 0) {
            return 0.0;
        }
    }
    return tree[nextId].content;
}

/**
 * sets the model count which corresponds to the given id
 *
 * @param id
 *      the id for which the model count should be set
 * @param tree
 *      a pointer to the tree structure
 * @param numVars
 *      the number of variables in the bag
 * @param treeSize
 *      the number of nodes in the tree
 * @param value
 *      the new value of the id
 */
__device__ void setCount(uint64_t id, TreeNode *tree, size_t numVars, uint64_t* treeSize, double value) {
    ulong nextId = 0;
    ulong val = 0;
    if (numVars == 0) {
        atomicAdd(treeSize, 1);
    }
    for (ulong i = 0; i < numVars; i++) { 
        // lower or upper 32bit, depending on if bit of variable i is set in id
        uint * lowVal = &((uint *) &(tree[nextId]))[(id >> (numVars - i - 1)) & 1];
        // secure our slot by incrementing treeSize
        if (val == 0 && *lowVal == 0) {
            val = atomicAdd(treeSize, 1) + 1;
        }
        atomicCAS(lowVal, 0, val);
        if (*lowVal == val) {
            if (i < (numVars - 1)) {
                val = atomicAdd(treeSize, 1) + 1;
            }
        }
        nextId = *lowVal;
    }
    tree[nextId].content = value;
}

/**
 * converts a array structure into a tree
 *
 * @param numVars
 *      the number of variables in the bag
 * @param tree
 *      a pointer to the tree structure
 * @param solutions_old
 *      array containing the models
 * @param treeSize
 *      the number of nodes in the tree
 * @param startId
  *     the start id of the current node
 * @param exponent
  *     the max exponent of this run
 */
__global__ void array2tree(
        size_t numVars, 
        TreeNode *tree, 
        const double *solutions_old,
        uint64_t *treeSize,
        int64_t startId,
        int64_t *exponent,
        const RunMeta meta
) {
    int64_t id = get_global_id(meta);
    if (id < 0) {
        return;
    }
    if (solutions_old[id] > 0) {
        setCount(id + startId, tree, numVars, treeSize, solutions_old[id]);
        if (!(meta.mode & NO_EXP)) {
            atomicMax(exponent, (int64_t)ilogb(solutions_old[id]));
        }
    }
}

/**
 * combines two tree structure into one
 *
 * @param numVars
 *      the number of variables in the bag
 * @param tree
 *      a pointer to the tree structure which will receive all the models from the other tree
 * @param solutions_old
 *      a pointer to the old tree structure
 * @param treeSize
 *      the number of nodes in the tree
 * @param startId
  *     the start id of the current node
 */
__global__ void combineTree(
        uint64_t numVars,
        TreeNode *tree,
        const TreeNode *solutions_old,
        uint64_t *treeSize,
        int64_t startId,
        const RunMeta meta
) {
    int64_t id = get_global_id(meta);
    if (id < 0) {
        return;
    }
    double val = getCount(id + startId, solutions_old, numVars);
    if (val > 0) {
        setCount(id + startId, tree, numVars, treeSize, val);
    }
}

/**
 * Operation to solve a Introduce node in the decomposition.
 *
 * @param variables
 *      the ids of the variables in the current bag
 * @param edge
 *      the number of models for each assignment of the next bag
 * @param edgeVariables
 *      variables in the next bag
 * @param minId
 *      the start id of the last bag
 * @param maxId
 *      the end id of the last bag
 * @param weights
 *      the variables weights for weighted model counting
 * @param id
 *      the id for which the introduce should be solved
 * @return
 *      the model count
 */
__device__ double solveIntroduce_(
        GPUVars variables,
        const std::variant<TreeSolution, ArraySolution> &edge,
        GPUVars edgeVariables,
        double *weights,
        int64_t id,
        SolveMode mode
) {
    int64_t otherId = 0;
    int64_t a = 0, b = 0;
    double weight = 1.0;
    for (b = 0; b < edgeVariables.count && a < variables.count; b++) {
        while ((variables.vars[a] != edgeVariables.vars[b])) {
            a++;
        }

        otherId = otherId | (((id >> a) & 1) << b);
        a++;
    };

    //weighted model count
    if (weights != 0) {
        for (b = 0, a = 0; a < variables.count; a++) {
            if (edgeVariables.vars == 0 || (variables.vars[a] != edgeVariables.vars[b])) {
                weight *= weights[((id >> a) & 1) > 0 ? variables.vars[a] * 2 : variables.vars[a] * 2 + 1];
            }
            if (edgeVariables.vars != 0 && (variables.vars[a] == edgeVariables.vars[b]) && (b < (edgeVariables.count - 1))) {
                b++;
            }
        }
    }

    if (!dataEmpty(edge) && otherId >= minId(edge) && otherId < maxId(edge)) {
        if (auto sol = std::get_if<TreeSolution>(&edge)) {
            return getCount(otherId, sol->tree, edgeVariables.count) * weight;
        } else if (auto sol = std::get_if<ArraySolution>(&edge)) {
            return sol->elements[otherId - sol->minId] * weight;
        } else {
            return -1.0;
        }
    } else if (dataEmpty(edge) && otherId >= minId(edge) && otherId < maxId(edge)) {
        return 0.0;
    } else {
        return -1.0;
    }
}

/**
 * Operation to check if an assignment satisfies the clauses of a SAT formula.
 *
 * @param clauses
 *      the clauses in the SAT formula
 * @param numVarsC
 *      array containing the number of Variables in each clause
 * @param numclauses
 *      the number of clauses in the sat formula
 * @param id
 *      the id of the thread - used to get the variable assignment
 * @param variables
 *      a vector containing the ids of the variables
 * @return
 *      1 - if the assignment satisfies the formula
 *      0 - if the assignment doesn't satisfy the formula
 */
__device__ int checkBag(long *clauses, long *numVarsC, long numclauses, int64_t id, GPUVars variables) {
    long i, varNum = 0;
    long satC = 0, a, b;
    // iterate through all clauses
    for (i = 0; i < numclauses; i++) {
        satC = 0;
        // iterate through clause variables
        for (a = 0; a < numVarsC[i] && !satC; a++) {
            satC = 1;
            //check current variables
            for (b = 0; b < variables.count; b++) {
                // check if clause is satisfied
                if ((clauses[varNum + a] == variables.vars[b]) ||
                    (clauses[varNum + a] == -variables.vars[b])) {
                    satC = 0;
                    if (clauses[varNum + a] < 0) {
                        //clause contains negative var and var is assigned negative
                        if ((id & (1 << (b))) == 0) {
                            satC = 1;
                            break;
                        }
                    } else {
                        //clause contains positive var and var is assigned positive
                        if ((id & (1 << (b))) > 0) {
                            satC = 1;
                            break;
                        }
                    }
                }
            }
        }
        varNum += numVarsC[i];
        // we have an unsattisifed clause
        if (!satC) {
            return 0;
        }
    }
    return 1;
}

/**
 * Operation to solve a Join node in the decomposition.
 *
 * @param solutions
 *      the number of solutions of the join
 * @param edge1
 *      contains the number of solutions in the first edge
 * @param edge2
 *      contains the number of solutions in the second edge
 * @param variables
 *      the variables in the join bag
 * @param edgeVariables1
 *      the variables in the bag of the first edge
 * @param edgeVariables2
 *      the variables in the bag of the second edge
 * @param numV
 *      the number of variables in the join bag
  * @param minId1
  *     the start id of the first edge
  * @param maxId1
  *     the end id of the first edge
  * @param minId2
  *     the start id of the second edge
  * @param maxId2
  *     the end id of the second edge
  * @param startIDNode
  *     the start id of the current node
  * @param weights
  *     the variable weights for weighted model counting
  * @param sols
  *     the number of assignments which lead to a solution
  * @param value
  *     correction value for the exponents
  * @param exponent
  *     the max exponent of this run
  */
__global__ void solveJoin(
        double* solutions,
        const std::optional<std::variant<TreeSolution, ArraySolution>> edge1,
        const std::optional<std::variant<TreeSolution, ArraySolution>> edge2,
        GPUVars variables,
        GPUVars edgeVariables1,
        GPUVars edgeVariables2,
        int64_t startIDNode,
        int64_t startIDEdge1,
        int64_t startIDEdge2,
        double *weights,
        uint64_t *sols,
        double value,
        int64_t *exponent,
        const RunMeta run
) {
    int64_t id = get_global_id(run);
    if (id < 0) {
        return;
    }

    double tmp = -1, tmp_ = -1;
    double weight = 1;
    if (edge1.has_value()) {
        // get solution count from first edge
        tmp = solveIntroduce_(variables, edge1.value(), edgeVariables1, weights, id, run.mode);
    }
    if (edge2.has_value()) {
        // get solution count from second edge
        tmp_ = solveIntroduce_(variables, edge2.value(), edgeVariables2, weights, id, run.mode);
    }
    // weighted model count
    if (weights != 0) {
        for (long a = 0; a < variables.count; a++) {
            weight *= weights[((id >> a) & 1) > 0 ? variables.vars[a] * 2 : variables.vars[a] * 2 + 1];
        }
    }

    if (tmp_ >= 0.0 && tmp >= 0.0) {
        if (tmp_ > 0.0 && tmp > 0.0) {
            atomicAdd(sols, 1);
        }
        if (!(run.mode & NO_EXP)) {
            solutions[id - startIDNode] = tmp_ * tmp / value / weight;
        } else {
            solutions[id - startIDNode] = tmp_ * tmp / weight;
        }
    }

        // we have some solutions in edge1
    else if (tmp >= 0.0) {
        double oldVal = solutions[id - startIDNode];
        if (oldVal < 0) {
            if (tmp > 0) {
                atomicAdd(sols, 1);
            }
        } else if (oldVal > 0) {
            if (tmp == 0) {
                atomicSub(sols, 1);
            }
        }
        if (oldVal < 0) {
            oldVal = 1.0;
        }
        solutions[id - startIDNode] = tmp * oldVal / weight;
    }

        // we have some solutions in edge2
    else if (tmp_ >= 0.0) {
        double oldVal = solutions[id - startIDNode];
        if (oldVal < 0) {
            if (tmp_ > 0) {
                atomicAdd(sols, 1);
            }
        } else if (oldVal > 0) {
            if (tmp_ == 0) {
                atomicSub(sols, 1);
            }
        }
        if (oldVal < 0) {
            oldVal = 1.0;
        }

        if (!(run.mode & NO_EXP)) {
            solutions[id - startIDNode] = tmp_ * oldVal / value;
        } else {
            solutions[id - startIDNode] = tmp_ * oldVal;
        }
    }
    if (run.mode & ARRAY_TYPE && !(run.mode & NO_EXP)) {
        atomicMax(exponent, ilogb(solutions[id - startIDNode]));
    }
}

/**
 * Operation to solve an Introduce
 *
 * @param clauses
 *      the clauses in the sat formula
 * @param numVarsC
 *      the number of variables for each clause
 * @param numclauses
 *      the number of clauses
 * @param variables
 *      the ids of the variables in the current bag
 * @param edge
 *      the number of models for each assignment of the last bags
 * @param edgeVariables
 *      variables of the last bag
 * @param minId
 *      the start id of the last bag
 * @param maxId
 *      the end id of the last bag
 * @param weights
 *      the variables weights for weighted model counting
 * @param id
 *      the id for which the introduce should be solved
 * @return
 *      the model count
 */
__device__ double solveIntroduceF(
        long *clauses,
        long *numVarsC,
        long numclauses,
        GPUVars variables,
        const std::variant<TreeSolution, ArraySolution> &edge,
        GPUVars edgeVariables,
        double *weights,
        long id,
        SolveMode mode
) {
    double tmp;
    if (!dataEmpty(edge)) {
        // get solutions count edge
        tmp = solveIntroduce_(variables, edge, edgeVariables, weights, id, mode);
    } else {
        // no edge - solve leaf
        tmp = 1.0;

        //weighted model count
        if (weights != 0) {
            for (long i = 0; i < variables.count; i++) {
                tmp *= weights[((id >> i) & 1) > 0 ? variables.vars[i] * 2 : variables.vars[i] * 2 + 1];
            }
        }
    }
    if (tmp > 0.0) {
        // check if assignment satisfies the given clauses
        int sat = checkBag(clauses, numVarsC, numclauses, id, variables);
        if (sat != 1) {
            return 0.0;
        } else {
            return tmp;
        }
    } else {
        return 0.0;
    }
}

/**
 * Operation to solve a Introduce and Forget node in the decomposition.
 *
 * @param solsF
 *      the number of models for each assignment
 * @param varsForget
 *      the variables after the forget operation
 * @param solsE
 *      the solutions from the last node
 * @param lastVars
 *      the variables from the alst oepration
 * @param combinations
 *      the number of assignments for which we have to collect the model counts
 * @param minIdE
 *      start id of the chunk from the last node
 * @param maxIdE
 *      end id of the chunk from the last node
 * @param startIDF
 *      start id of the chung from the current node
 * @param sols
  *     the number of assignments which lead to a solution
 * @param varsI
 *      the variables after the introduce
 * @param clauses
 *      the clauses which only contain variables from the introduce operation
 * @param numVarsC
 *      the number of variables per clause
 * @param numclauses
 *      the number of clauses
 * @param weights
 *      the variables weights for weighted model counting
 * @param exponent
  *     the max exponent of this run
 * @param value
  *     correction value for the exponents
 */
__global__ void solveIntroduceForget(
        const std::variant<TreeSolution, ArraySolution> solsF,
        GPUVars varsForget,
        const std::variant<TreeSolution, ArraySolution> solsE,
        GPUVars lastVars,
        uint64_t combinations,
        int64_t startIDF,
        int64_t startIDE,
        uint64_t *sols,
        GPUVars varsIntroduce,
        long *clauses,
        long *numVarsC,
        long numclauses,
        double *weights,
        long *exponent,
        double value,
        const RunMeta run
) {
    int64_t id = get_global_id(run);
    if (id < 0) {
        return;
    }
    if (varsIntroduce.count != varsForget.count) {
        double tmp = 0;
        long templateId = 0;
        // generate templateId
        for (long i = 0, a = 0; i < varsIntroduce.count && a < varsForget.count; i++) {
            if (varsIntroduce.vars[i] == varsForget.vars[a]) {
                templateId = templateId | (((id >> a) & 1) << i);
                a++;
            }
        }

        // iterate though all corresponding edge solutions
        for (long i = 0; i < combinations; i++) {
            long b = 0, otherId = templateId;
            for (long a = 0; a < varsIntroduce.count; a++) {
                if (b >= varsForget.count || varsIntroduce.vars[a] != varsForget.vars[b]) {
                    otherId = otherId | (((i >> (a - b)) & 1) << a);
                } else {
                    b++;
                }
            }
            tmp += solveIntroduceF(clauses, numVarsC, numclauses, varsIntroduce, solsE, lastVars, weights, otherId, run.mode);
        }
        
        if (tmp > 0) {
            if (auto sol = std::get_if<TreeSolution>(&solsF)) {
                double last = getCount(id, sol->tree, varsForget.count);
                if (!(run.mode & NO_EXP))  {
                    setCount(id, sol->tree, varsForget.count, sols, (tmp / value + last));
                    atomicMax(exponent, ilogb((tmp / value + last)));
                } else {
                    setCount(id, sol->tree, varsForget.count, sols, (tmp + last));
                }
            } else if (auto sol = std::get_if<ArraySolution>(&solsF)) {
                double last=sol->elements[id - (startIDF)];
                atomicAdd(sols, 1);
                //*sols += 1;
                if (!(run.mode & NO_EXP))  {
                    sol->elements[id - (startIDF)] = tmp / value + last;
                    atomicMax(exponent, ilogb(tmp / value + last));
                } else {
                    sol->elements[id - (startIDF)] = tmp + last;
                }
            }
        }
    } else {
        // no forget variables, only introduce
        double tmp = solveIntroduceF(clauses, numVarsC, numclauses, varsIntroduce, solsE, lastVars, weights, id, run.mode);
        if (tmp > 0) {
            if (auto sol = std::get_if<TreeSolution>(&solsF)) {
                double last = getCount(id, sol->tree, varsForget.count);
                if (!(run.mode & NO_EXP))  {
                    setCount(id, sol->tree, varsForget.count, sols, (tmp / value + last));
                    atomicMax(exponent, ilogb((tmp / value + last)));
                } else {
                    setCount(id, sol->tree, varsForget.count, sols, (tmp + last));
                }
            } else if (auto sol = std::get_if<ArraySolution>(&solsF)) {
                double last=sol->elements[id - (startIDF)];
                atomicAdd(sols, 1);
                //*sols += 1;
                if (!(run.mode & NO_EXP))  {
                    sol->elements[id - (startIDF)] = tmp / value + last;
                    atomicMax(exponent, ilogb(tmp / value + last));
                } else {
                    sol->elements[id - (startIDF)] = tmp + last;
                }
            }
        }
    } 
}

void combineTreeWrapper(
    uint64_t numVars,
    TreeNode *tree,
    const TreeNode *solutions_old,
    uint64_t *treeSize,
    int64_t startId,
    RunMeta meta
) {
    int64_t threadsPerBlock = 512;
    int64_t threads = meta.maxId - meta.minId;
    int64_t blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    combineTree<<<blocksPerGrid, threadsPerBlock>>>(
        numVars,
        tree,
        solutions_old,
        treeSize,
        startId,
        meta
    );
    gpuErrchk(hipDeviceSynchronize());
}

void array2treeWrapper(
    size_t numVars, 
    TreeNode *tree, 
    const double *solutions_old,
    uint64_t *treeSize,
    int64_t startId,
    int64_t *exponent,
    RunMeta meta
) {
    
    int64_t threadsPerBlock = 512;
    int64_t threads = meta.maxId - meta.minId;
    int64_t blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    array2tree<<<blocksPerGrid, threadsPerBlock>>>(
        numVars,
        tree,
        solutions_old,
        treeSize,
        startId,
        exponent,
        meta
    );
    gpuErrchk(hipDeviceSynchronize());
}

void solveJoinWrapper(
    double *solutions,
    std::optional<std::variant<TreeSolution, ArraySolution>> edge1,
    std::optional<std::variant<TreeSolution, ArraySolution>> edge2,
    GPUVars variables,
    GPUVars edgeVariables1,
    GPUVars edgeVariables2,
    int64_t startIDNode,
    int64_t startIDEdge1,
    int64_t startIDEdge2,
    double *weights,
    uint64_t *sols,
    double value,
    int64_t *exponent,
    RunMeta meta
) {
    int64_t threadsPerBlock = 512;
    int64_t threads = meta.maxId - meta.minId;
    int64_t blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    solveJoin<<<blocksPerGrid, threadsPerBlock>>>(
        solutions,
        std::move(edge1),
        std::move(edge2),
        variables,
        edgeVariables1,
        edgeVariables2,
        startIDNode,
        startIDEdge1,
        startIDEdge2,
        weights,
        sols,
        value, 
        exponent,
        meta
    );

    gpuErrchk(hipDeviceSynchronize());
}

void introduceForgetWrapper(
    std::variant<TreeSolution, ArraySolution> solsF,
    GPUVars varsForget,
    std::variant<TreeSolution, ArraySolution> solsE,
    GPUVars lastVars,
    uint64_t combinations,
    int64_t startIDF,
    int64_t startIDE,
    uint64_t *sols,
    GPUVars varsIntroduce,
    long *clauses,
    long *numVarsC,
    long numclauses,
    double *weights,
    int64_t *exponent,
    double value,
    RunMeta meta
) {
    int64_t threadsPerBlock = 512;
    int64_t threads = meta.maxId - meta.minId;
    int64_t blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    solveIntroduceForget<<<blocksPerGrid, threadsPerBlock>>>(
        std::move(solsF),
        varsForget,
        std::move(solsE),
        lastVars,
        combinations,
        startIDF,
        startIDE,
        sols,
        varsIntroduce,
        clauses,
        numVarsC,
        numclauses,
        weights,
        exponent, 
        value,
        meta
    );

    gpuErrchk(hipDeviceSynchronize());
}
}
