#include "hip/hip_runtime.h"
#define GPU_HOST_ATTR __device__ __host__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <types.h>



using namespace gpusat;


__device__ long atomicAdd(long* address, long val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        (val +
                               (long)(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return old;
}

__device__ long atomicSub(long* address, long val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        ((long)(assumed) - val));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return old;
}


__device__ long atomicMax(long* address, long val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        max(val,
                               (long)(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return old;
}


// FIXME: normal atomicAdd might not be atomic across devices


__device__ long get_global_id() {
    // TODO: y and z
    return blockDim.x * blockIdx.x + threadIdx.x;
}

/**
 * returns the model count which corresponds to the given id
 *
 * @param id
 *      the id for which the model count should be returned
 * @param tree
 *      a pointer to the tree structure
 * @param numVars
 *      the number of variables in the bag
 * @return
 *      the model count
 */
__device__ double getCount(long id, const TreeNode *tree, long numVars) {
    ulong nextId = 0;
    for (ulong i = 0; i < numVars; i++) {
        nextId = ((uint *) &(tree[nextId]))[(id >> (numVars - i - 1)) & 1];
        if (nextId == 0) {
            return 0.0;
        }
    }
    return tree[nextId].content;
}

/**
 * sets the model count which corresponds to the given id
 *
 * @param id
 *      the id for which the model count should be set
 * @param tree
 *      a pointer to the tree structure
 * @param numVars
 *      the number of variables in the bag
 * @param treeSize
 *      the number of nodes in the tree
 * @param value
 *      the new value of the id
 */
__device__ void setCount(long id, long *tree, long numVars, long *treeSize, double value) {
    ulong nextId = 0;
    ulong val = 0;
    if (numVars == 0) {
        atomicAdd(treeSize, 1);
    }
    for (ulong i = 0; i < numVars; i++) { 
        // lower or upper 32bit, depending on if bit of variable i is set in id
        uint * lowVal = &((uint *) &(tree[nextId]))[(id >> (numVars - i - 1)) & 1];
        // secure our slot by incrementing treeSize
        if (val == 0 && *lowVal == 0) {
            val = atomicAdd(treeSize, 1) + 1;
        }
        atomicCAS(lowVal, 0, val);
        if (*lowVal == val) {
            if (i < (numVars - 1)) {
                val = atomicAdd(treeSize, 1) + 1;
            }
        }
        nextId = *lowVal;
    }
    tree[nextId] = __double_as_longlong(value);
}

/**
 * converts a array structure into a tree
 *
 * @param numVars
 *      the number of variables in the bag
 * @param tree
 *      a pointer to the tree structure
 * @param solutions_old
 *      array containing the models
 * @param treeSize
 *      the number of nodes in the tree
 * @param startId
  *     the start id of the current node
 * @param exponent
  *     the max exponent of this run
 */
__global__ void array2tree(long numVars, long *tree, double *solutions_old, long *treeSize, long startId, long *exponent, long id_offset, long max_id, SolveMode mode) {
    long id = get_global_id() + id_offset;
    if (id >= max_id) {
        return;
    }
    if (solutions_old[id] > 0) {
        setCount(id + startId, tree, numVars, treeSize, solutions_old[id]);
        if (!(mode & NO_EXP)) {
            atomicMax(exponent, ilogb(solutions_old[id]));
        }
    }
}

/**
 * combines two tree structure into one
 *
 * @param numVars
 *      the number of variables in the bag
 * @param tree
 *      a pointer to the tree structure which will receive all the models from the other tree
 * @param solutions_old
 *      a pointer to the old tree structure
 * @param treeSize
 *      the number of nodes in the tree
 * @param startId
  *     the start id of the current node
 */
__global__ void combineTree(long numVars, long *tree, long *solutions_old, long *treeSize, long startId, long id_offset, long max_id) {
    long id = get_global_id() + id_offset;
    if (id >= max_id) {
        return;
    }
    double val = getCount(id + startId, (TreeNode*)solutions_old, numVars);
    if (val > 0) {
        setCount(id + startId, tree, numVars, treeSize, val);
    }
}

/**
 * Operation to solve a Introduce node in the decomposition.
 *
 * @param variables
 *      the ids of the variables in the current bag
 * @param edge
 *      the number of models for each assignment of the next bag
 * @param edgeVariables
 *      variables in the next bag
 * @param minId
 *      the start id of the last bag
 * @param maxId
 *      the end id of the last bag
 * @param weights
 *      the variables weights for weighted model counting
 * @param id
 *      the id for which the introduce should be solved
 * @return
 *      the model count
 */
__device__ double solveIntroduce_(
        GPUVars variables,
        const std::variant<TreeSolution, ArraySolution> &edge,
        GPUVars edgeVariables,
        double *weights,
        long id,
        SolveMode mode
) {
    long otherId = 0;
    long a = 0, b = 0;
    double weight = 1.0;
    for (b = 0; b < edgeVariables.count && a < variables.count; b++) {
        while ((variables.vars[a] != edgeVariables.vars[b])) {
            a++;
        }

        otherId = otherId | (((id >> a) & 1) << b);
        a++;
    };

    //weighted model count
    if (weights != 0) {
        for (b = 0, a = 0; a < variables.count; a++) {
            if (edgeVariables.vars == 0 || (variables.vars[a] != edgeVariables.vars[b])) {
                weight *= weights[((id >> a) & 1) > 0 ? variables.vars[a] * 2 : variables.vars[a] * 2 + 1];
            }
            if (edgeVariables.vars != 0 && (variables.vars[a] == edgeVariables.vars[b]) && (b < (edgeVariables.count - 1))) {
                b++;
            }
        }
    }

    if (!dataEmpty(edge) && otherId >= minId(edge) && otherId < maxId(edge)) {
        if (auto sol = std::get_if<TreeSolution>(&edge)) {
            return getCount(otherId, sol->tree, edgeVariables.count) * weight;
        } else if (auto sol = std::get_if<ArraySolution>(&edge)) {
            return __longlong_as_double(sol->elements[otherId - sol->minId]) * weight;
        } else {
            return -1.0;
        }
    } else if (dataEmpty(edge) && otherId >= minId(edge) && otherId < maxId(edge)) {
        return 0.0;
    } else {
        return -1.0;
    }
}

/**
 * Operation to check if an assignment satisfies the clauses of a SAT formula.
 *
 * @param clauses
 *      the clauses in the SAT formula
 * @param numVarsC
 *      array containing the number of Variables in each clause
 * @param numclauses
 *      the number of clauses in the sat formula
 * @param id
 *      the id of the thread - used to get the variable assignment
 * @param variables
 *      a vector containing the ids of the variables
 * @return
 *      1 - if the assignment satisfies the formula
 *      0 - if the assignment doesn't satisfy the formula
 */
__device__ int checkBag(long *clauses, long *numVarsC, long numclauses, long id, GPUVars variables) {
    long i, varNum = 0;
    long satC = 0, a, b;
    // iterate through all clauses
    for (i = 0; i < numclauses; i++) {
        satC = 0;
        // iterate through clause variables
        for (a = 0; a < numVarsC[i] && !satC; a++) {
            satC = 1;
            //check current variables
            for (b = 0; b < variables.count; b++) {
                // check if clause is satisfied
                if ((clauses[varNum + a] == variables.vars[b]) ||
                    (clauses[varNum + a] == -variables.vars[b])) {
                    satC = 0;
                    if (clauses[varNum + a] < 0) {
                        //clause contains negative var and var is assigned negative
                        if ((id & (1 << (b))) == 0) {
                            satC = 1;
                            break;
                        }
                    } else {
                        //clause contains positive var and var is assigned positive
                        if ((id & (1 << (b))) > 0) {
                            satC = 1;
                            break;
                        }
                    }
                }
            }
        }
        varNum += numVarsC[i];
        // we have an unsattisifed clause
        if (!satC) {
            return 0;
        }
    }
    return 1;
}

/**
 * Operation to solve a Join node in the decomposition.
 *
 * @param solutions
 *      the number of solutions of the join
 * @param edge1
 *      contains the number of solutions in the first edge
 * @param edge2
 *      contains the number of solutions in the second edge
 * @param variables
 *      the variables in the join bag
 * @param edgeVariables1
 *      the variables in the bag of the first edge
 * @param edgeVariables2
 *      the variables in the bag of the second edge
 * @param numV
 *      the number of variables in the join bag
  * @param minId1
  *     the start id of the first edge
  * @param maxId1
  *     the end id of the first edge
  * @param minId2
  *     the start id of the second edge
  * @param maxId2
  *     the end id of the second edge
  * @param startIDNode
  *     the start id of the current node
  * @param weights
  *     the variable weights for weighted model counting
  * @param sols
  *     the number of assignments which lead to a solution
  * @param value
  *     correction value for the exponents
  * @param exponent
  *     the max exponent of this run
  */
__global__ void solveJoin(
        double* solutions,
        const std::optional<std::variant<TreeSolution, ArraySolution>> edge1,
        const std::optional<std::variant<TreeSolution, ArraySolution>> edge2,
        GPUVars variables,
        GPUVars edgeVariables1,
        GPUVars edgeVariables2,
        long startIDNode,
        long startIDEdge1,
        long startIDEdge2,
        double *weights,
        long *sols,
        double value,
        long *exponent,
        long id_offset,
        long max_id,
        SolveMode mode
) {
    long id = get_global_id() + id_offset;
    if (id >= max_id) {
        return;
    }

    double tmp = -1, tmp_ = -1;
    double weight = 1;
    if (edge1.has_value()) {
        // get solution count from first edge
        tmp = solveIntroduce_(variables, edge1.value(), edgeVariables1, weights, id, mode);
    }
    if (edge2.has_value()) {
        // get solution count from second edge
        tmp_ = solveIntroduce_(variables, edge2.value(), edgeVariables2, weights, id, mode);
    }
    // weighted model count
    if (weights != 0) {
        for (long a = 0; a < variables.count; a++) {
            weight *= weights[((id >> a) & 1) > 0 ? variables.vars[a] * 2 : variables.vars[a] * 2 + 1];
        }
    }


    if (tmp_ >= 0.0 && tmp >= 0.0) {
        if (tmp_ > 0.0 && tmp > 0.0) {
            atomicAdd(sols, 1);
        }
        if (!(mode & NO_EXP)) {
            solutions[id - startIDNode] = tmp_ * tmp / value / weight;
        } else {
            solutions[id - startIDNode] = tmp_ * tmp / weight;
        }
    }

        // we have some solutions in edge1
    else if (tmp >= 0.0) {
        double oldVal = solutions[id - startIDNode];
        if (oldVal < 0) {
            if (tmp > 0) {
                atomicAdd(sols, 1);
            }
        } else if (oldVal > 0) {
            if (tmp == 0) {
                atomicSub(sols, 1);
            }
        }
        if (oldVal < 0) {
            oldVal = 1.0;
        }
        solutions[id - startIDNode] = tmp * oldVal / weight;
    }

        // we have some solutions in edge2
    else if (tmp_ >= 0.0) {
        double oldVal = solutions[id - startIDNode];
        if (oldVal < 0) {
            if (tmp_ > 0) {
                atomicAdd(sols, 1);
            }
        } else if (oldVal > 0) {
            if (tmp_ == 0) {
                atomicSub(sols, 1);
            }
        }
        if (oldVal < 0) {
            oldVal = 1.0;
        }

        if (!(mode & NO_EXP)) {
            solutions[id - startIDNode] = tmp_ * oldVal / value;
        } else {
            solutions[id - startIDNode] = tmp_ * oldVal;
        }
    }
    if (mode & ARRAY_TYPE && !(mode & NO_EXP)) {
        atomicMax(exponent, ilogb(__longlong_as_double(solutions[id - startIDNode])));
    }
}

/**
 * Operation to solve an Introduce
 *
 * @param clauses
 *      the clauses in the sat formula
 * @param numVarsC
 *      the number of variables for each clause
 * @param numclauses
 *      the number of clauses
 * @param variables
 *      the ids of the variables in the current bag
 * @param edge
 *      the number of models for each assignment of the last bags
 * @param edgeVariables
 *      variables of the last bag
 * @param minId
 *      the start id of the last bag
 * @param maxId
 *      the end id of the last bag
 * @param weights
 *      the variables weights for weighted model counting
 * @param id
 *      the id for which the introduce should be solved
 * @return
 *      the model count
 */
__device__ double solveIntroduceF(
        long *clauses,
        long *numVarsC,
        long numclauses,
        GPUVars variables,
        const std::variant<TreeSolution, ArraySolution> &edge,
        GPUVars edgeVariables,
        double *weights,
        long id,
        SolveMode mode
) {
    double tmp;
    if (!dataEmpty(edge)) {
        // get solutions count edge
        tmp = solveIntroduce_(variables, edge, edgeVariables, weights, id, mode);
    } else {
        // no edge - solve leaf
        tmp = 1.0;

        //weighted model count
        if (weights != 0) {
            for (long i = 0; i < variables.count; i++) {
                tmp *= weights[((id >> i) & 1) > 0 ? variables.vars[i] * 2 : variables.vars[i] * 2 + 1];
            }
        }
    }
    if (tmp > 0.0) {
        // check if assignment satisfies the given clauses
        int sat = checkBag(clauses, numVarsC, numclauses, id, variables);
        if (sat != 1) {
            return 0.0;
        } else {
            return tmp;
        }
    } else {
        return 0.0;
    }
}

/**
 * Operation to solve a Introduce and Forget node in the decomposition.
 *
 * @param solsF
 *      the number of models for each assignment
 * @param varsForget
 *      the variables after the forget operation
 * @param solsE
 *      the solutions from the last node
 * @param lastVars
 *      the variables from the alst oepration
 * @param combinations
 *      the number of assignments for which we have to collect the model counts
 * @param minIdE
 *      start id of the chunk from the last node
 * @param maxIdE
 *      end id of the chunk from the last node
 * @param startIDF
 *      start id of the chung from the current node
 * @param sols
  *     the number of assignments which lead to a solution
 * @param varsI
 *      the variables after the introduce
 * @param clauses
 *      the clauses which only contain variables from the introduce operation
 * @param numVarsC
 *      the number of variables per clause
 * @param numclauses
 *      the number of clauses
 * @param weights
 *      the variables weights for weighted model counting
 * @param exponent
  *     the max exponent of this run
 * @param value
  *     correction value for the exponents
 */
__global__ void solveIntroduceForget(
        const std::variant<TreeSolution, ArraySolution> solsF,
        GPUVars varsForget,
        const std::variant<TreeSolution, ArraySolution> solsE,
        GPUVars lastVars,
        long combinations,
        long startIDF,
        long startIDE,
        long *sols,
        GPUVars varsIntroduce,
        long *clauses,
        long *numVarsC,
        long numclauses,
        double *weights,
        long *exponent,
        double value,
        long id_offset,
        long max_id,
        SolveMode mode
) {
    long id = get_global_id() + id_offset;
    if (id >= max_id) {
        return;
    }
    if (varsIntroduce.count != varsForget.count) {
        double tmp = 0;
        long templateId = 0;
        // generate templateId
        for (long i = 0, a = 0; i < varsIntroduce.count && a < varsForget.count; i++) {
            if (varsIntroduce.vars[i] == varsForget.vars[a]) {
                templateId = templateId | (((id >> a) & 1) << i);
                a++;
            }
        }

        // iterate though all corresponding edge solutions
        for (long i = 0; i < combinations; i++) {
            long b = 0, otherId = templateId;
            for (long a = 0; a < varsIntroduce.count; a++) {
                if (b >= varsForget.count || varsIntroduce.vars[a] != varsForget.vars[b]) {
                    otherId = otherId | (((i >> (a - b)) & 1) << a);
                } else {
                    b++;
                }
            }
            tmp += solveIntroduceF(clauses, numVarsC, numclauses, varsIntroduce, solsE, lastVars, weights, otherId, mode);
        }
        
        if (tmp > 0) {
            if (auto sol = std::get_if<TreeSolution>(&solsF)) {
                double last = getCount(id, sol->tree, varsForget.count);
                if (!(mode & NO_EXP))  {
                    setCount(id, (long*)sol->tree, varsForget.count, sols, (tmp / value + last));
                    atomicMax(exponent, ilogb((tmp / value + last)));
                } else {
                    setCount(id, (long*)sol->tree, varsForget.count, sols, (tmp + last));
                }
            } else if (auto sol = std::get_if<ArraySolution>(&solsF)) {
                double last=__longlong_as_double(sol->elements[id - (startIDF)]);
                atomicAdd(sols, 1);
                //*sols += 1;
                if (!(mode & NO_EXP))  {
                    sol->elements[id - (startIDF)] = __double_as_longlong(tmp / value + last);
                    atomicMax(exponent, ilogb(tmp / value + last));
                } else {
                    sol->elements[id - (startIDF)] = __double_as_longlong(tmp + last);
                }
            }
        }
    } else {
        // no forget variables, only introduce
        double tmp = solveIntroduceF(clauses, numVarsC, numclauses, varsIntroduce, solsE, lastVars, weights, id, mode);
        if (tmp > 0) {
            if (auto sol = std::get_if<TreeSolution>(&solsF)) {
                double last = getCount(id, sol->tree, varsForget.count);
                if (!(mode & NO_EXP))  {
                    setCount(id, (long*)sol->tree, varsForget.count, sols, (tmp / value + last));
                    atomicMax(exponent, ilogb((tmp / value + last)));
                } else {
                    setCount(id, (long*)sol->tree, varsForget.count, sols, (tmp + last));
                }
            } else if (auto sol = std::get_if<ArraySolution>(&solsF)) {
                double last=__longlong_as_double(sol->elements[id - (startIDF)]);
                atomicAdd(sols, 1);
                //*sols += 1;
                if (!(mode & NO_EXP))  {
                    sol->elements[id - (startIDF)] = __double_as_longlong(tmp / value + last);
                    atomicMax(exponent, ilogb(tmp / value + last));
                } else {
                    sol->elements[id - (startIDF)] = __double_as_longlong(tmp + last);
                }
            }
        }
    }
}


__global__ void helloWorldKernel(int val)
{
    printf("[%d, %d]:\t\tHello, World! Val: %d\n",\
            blockIdx.y*gridDim.x+blockIdx.x,\
            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x, val);
}

void combineTreeWrapper(long numVars, long *tree, long *solutions_old, long *treeSize, long startId, size_t threads, long id_offset) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    combineTree<<<blocksPerGrid, threadsPerBlock>>>(
        numVars,
        tree,
        solutions_old,
        treeSize,
        startId,
        id_offset,
        threads + id_offset
    );
    hipDeviceSynchronize();
}

void array2treeWrapper(long numVars, long *tree, double *solutions_old, long *treeSize, long startId, long *exponent, size_t threads, long id_offset, SolveMode mode) {
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    array2tree<<<blocksPerGrid, threadsPerBlock>>>(
        numVars,
        tree,
        solutions_old,
        treeSize,
        startId,
        exponent,
        id_offset,
        threads + id_offset,
        mode
    );
    hipDeviceSynchronize();
}

void solveJoinWrapper(
        double *solutions,
        std::optional<std::variant<TreeSolution, ArraySolution>> edge1,
        std::optional<std::variant<TreeSolution, ArraySolution>> edge2,
        GPUVars variables,
        GPUVars edgeVariables1,
        GPUVars edgeVariables2,
        long startIDNode,
        long startIDEdge1,
        long startIDEdge2,
        double *weights,
        long *sols,
        double value,
        long *exponent,
        size_t threads,
        long id_offset,
        SolveMode mode
) {

    int threadsPerBlock = 256;
    int blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    solveJoin<<<blocksPerGrid, threadsPerBlock>>>(
                    solutions,
                    std::move(edge1),
                    std::move(edge2),
                    variables,
                    edgeVariables1,
                    edgeVariables2,
                    startIDNode,
                    startIDEdge1,
                    startIDEdge2,
                    weights,
                    sols,
                    value, 
                    exponent,
                    id_offset,
                    threads + id_offset,
                    mode
    );

    hipDeviceSynchronize();
}

void introduceForgetWrapper(
        std::variant<TreeSolution, ArraySolution> solsF,
        GPUVars varsForget,
        std::variant<TreeSolution, ArraySolution> solsE,
        GPUVars lastVars,
        long combinations,
        long startIDF,
        long startIDE,
        long *sols,
        GPUVars varsIntroduce,
        long *clauses,
        long *numVarsC,
        long numclauses,
        double *weights,
        long *exponent,
        double value,
        size_t threads,
        long id_offset,
        SolveMode mode
    ) {

    int threadsPerBlock = 256;
    int blocksPerGrid = (threads + threadsPerBlock - 1) / threadsPerBlock;
    solveIntroduceForget<<<blocksPerGrid, threadsPerBlock>>>(
                    std::move(solsF),
                    varsForget,
                    std::move(solsE),
                    lastVars,
                    combinations,
                    startIDF,
                    startIDE,
                    sols,
                    varsIntroduce,
                    clauses,
                    numVarsC,
                    numclauses,
                    weights,
                    exponent, 
                    value,
                    id_offset,
                    threads + id_offset,
                    mode);

    hipDeviceSynchronize();
    /*
    int *mem;
    dim3 threads(2, 1);
    dim3 blocks(1, 1);

    hipMalloc((void**)&mem, sizeof(int));

    hipMemcpy(mem, &val, sizeof(int), hipMemcpyHostToDevice);

    printf("running kernel..\n");
    helloWorldKernel<<< blocks, threads>>>(mem);

    printf("synchronize: %d\n", hipDeviceSynchronize());
    hipFree(mem);
    */
}
